#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "TAFPRAY2D.h"
// includes CUDA

#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> 
#include <hip/hip_vector_types.h>

#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <driver_functions.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h> 

typedef unsigned int  uint;

// Volume size setting
int2 volumeSize;
hipArray *d_volumeArray = 0;
float *cudavolume;

//Table size setting
int2 tableSize;
hipArray *d_tableArray = 0;
float *cudaTable;

//GeoDiv size setting
int2 geodivSize;
hipArray *d_geodivArray = 0;

//Line size setting
float4 *cudaLines;

// 2D texture for projection
texture<float, hipTextureType2D, hipReadModeElementType> texVolume;
texture<float, hipTextureType2D, hipReadModeElementType> areaTex;
texture<float, hipTextureType2D, hipReadModeElementType> geodivTex;
texture<float4, hipTextureType1D, hipReadModeElementType> lineTex;

//CUDA  constant parameters definition
__constant__  int   c_nx;
__constant__  int   c_ny;
__constant__  float c_dx;
__constant__  float c_dy;
__constant__  float c_offset_x;
__constant__  float c_offset_y;
__constant__  float c_dso;
__constant__  float c_dsd;
__constant__  int   c_nr;
__constant__  float c_dr;
__constant__  float c_offset_r;
__constant__  float c_angle_start;
__constant__  int   c_angle_num;
__constant__  int   c_TaDimx;
__constant__  int   c_TaDimy;
__constant__  float c_TaDeltax;
__constant__  float c_TaDeltay;
__constant__  float c_rr;
__constant__  float c_xx;
__constant__  float c_yy;
__constant__  int c_nfoot;
__constant__ float2 c_alutOffset;
__constant__ float  c_voxBase;
__constant__ int    c_nlines;
__constant__ float2 c_src;
__constant__ float2 c_uv_s;
__constant__ float2 c_uv_t;
__constant__ float  c_viewang;

struct Ray
{
	float2  o;  // ray origin site
	float2  d;  // ray  direction
};
// help functions
int iDivUp(int a, int b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

//rotate around Z 
float2 rotateCCW_z(float2 v, float cs, float sn)
{
	return make_float2(v.x*cs - v.y*sn, v.x*sn + v.y*cs);
}

extern "C"
void InitProjection()
{
	//create 2D array
	hipChannelFormatDesc chaDesc1 = hipCreateChannelDesc<float>();

	// allocate 2D array to bind 2D texture in cuda
	hipMallocArray(&d_volumeArray, &chaDesc1, volumeSize.x, volumeSize.y);

}

extern "C"
void BindImgToTex2D()
{

	hipChannelFormatDesc channelDescVolume = hipCreateChannelDesc<float>();

	// copy data to 2D array
	hipMemcpyToArray(d_volumeArray, 0, 0, cudavolume, volumeSize.x * volumeSize.y * sizeof(float), hipMemcpyDeviceToDevice);

	// set texture parameters
	texVolume.normalized = false;
	texVolume.filterMode = hipFilterModePoint;
	texVolume.addressMode[0] = hipAddressModeClamp;
	texVolume.addressMode[1] = hipAddressModeClamp;

	// bind texture array to 2D volume
	hipBindTextureToArray(texVolume, d_volumeArray, channelDescVolume);
}

extern "C"
void InitGeodiv()
{
	hipChannelFormatDesc chaDesc1 = hipCreateChannelDesc<float>();

	// allocate 2D array to bind 2D texture in cuda
	hipMallocArray(&d_geodivArray, &chaDesc1, geodivSize.x, geodivSize.y);
}

extern "C"
void BindGeodiv(float* Geodiv)
{
	hipChannelFormatDesc channelDescGeo = hipCreateChannelDesc<float>();

	// copy data to 2D array
	hipMemcpyToArray(d_geodivArray, 0, 0, Geodiv, geodivSize.x * geodivSize.y * sizeof(float), hipMemcpyHostToDevice);

	geodivTex.addressMode[0] = hipAddressModeClamp;
	geodivTex.addressMode[1] = hipAddressModeClamp;
	geodivTex.filterMode = hipFilterModePoint;
	geodivTex.normalized = false;

	// bind texture array to 2D Geodiv
	hipBindTextureToArray(geodivTex, d_geodivArray, channelDescGeo);

}

/////////////
////////////CUDA kernel functions
__device__  static float fetchAreaLut(int sidx, float2	xy)
{
	sidx = clamp(sidx, 0, c_nlines - 1);
	float4 L = tex1Dfetch(lineTex, sidx);
	float  ang, pos;
	ang = L.x,
		pos = L.y*xy.x + L.z*xy.y + L.w;

	float value;
	value = tex2D(areaTex,
		fabsf(pos)*c_alutOffset.x + 0.5f,
		ang*c_alutOffset.y + 0.5f);

	return pos < 0.0f ? c_voxBase - value : value;

}

__global__ static void update_lines_kernel(float4 *lines, float beta)
{
	int is = blockIdx.x*blockDim.x + threadIdx.x;
	if (is >= c_nlines)
		return;

	float s0 = -c_rr + c_offset_r*c_dr;
	float gamma = s0 + is*c_dr;

	float2 P1 = c_src + make_float2(c_dsd*sin(beta + gamma), -c_dsd*cos(beta + gamma));
	float2 rayvec = P1 - c_src;

	float ang = atan2f(rayvec.y, rayvec.x) * (360.0f / (2.0f*(float)M_PI));
	if (ang < 0.0f)
		ang += 360.0f;

	float A = P1.y - c_src.y;
	float B = c_src.x - P1.x;
	float C = P1.x*c_src.y - c_src.x*P1.y;
	float Z = sqrtf(A*A + B*B);

	if (ang <= 45.0f) { ; }
	else if (ang <= 90.0f) { ang = 90.0f - ang; }
	else if (ang <= 135.0f) { ang = ang - 90.0f; }
	else if (ang <= 180.0f) { ang = 180.0f - ang; }
	else if (ang <= 225.0f) { ang = ang - 180.0f; }
	else if (ang <= 270.0f) { ang = 270.0f - ang; }
	else if (ang <= 315.0f) { ang = ang - 270.0f; }
	else { ang = 360.0f - ang; }

	lines[is] = make_float4(ang, A / Z, B / Z, C / Z);

}

__global__ static void  lut_init_foot_kernel(float* footinfo)
{
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	if (ix >= c_nx || iy >= c_ny)
		return;

	int stride = c_nx*c_ny;
	footinfo += (iy*c_nx + ix);

	int nfoot2 = c_nfoot / 2;
	float2 xy = make_float2(((ix + 0.5) * c_dx - c_xx*1.0f + c_offset_x),
		((iy + 0.5) * c_dy - c_yy*1.0f + c_offset_y));

	float div = sqrt(SQR(xy.x - c_src.x) + SQR(xy.y - c_src.y));
	*footinfo = div; footinfo += stride;

	float mag = c_dsd / sqrt(SQR(dot(c_uv_s, xy)) + SQR((dot(c_uv_t, xy) + c_dso)));
	*footinfo = mag;
	footinfo += stride;

	float gamma = atan(dot(c_uv_s, xy) / (dot(c_uv_t, xy) + c_dso));
	int   s_bin = (int)floorf(gamma / c_dr + 0.5f*(c_nr - 1) - c_offset_r) - nfoot2;

	*footinfo = float(s_bin);
	footinfo += stride;

	int is = s_bin;
	float area0 = fetchAreaLut(is, xy); ++is;
	for (int ifoot = 0; ifoot < c_nfoot; ++ifoot, ++is, footinfo += stride)
	{
		float area1 = fetchAreaLut(is, xy);
		*footinfo = fabsf(area0 - area1);
		area0 = area1;
	}

}

__global__ static void lut_fp_kernel(float *d_proj, float *footinfo, const float val)
{
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	int iy = blockIdx.y*blockDim.y + threadIdx.y;

	if (ix >= c_nx || iy >= c_ny)
		return;

	int stride = c_nx*c_ny;
	footinfo += (iy*c_nx + ix);

	float shdata[8];
	float* p_div = shdata + 0;
	float* p_sbin = shdata + 2;
	float* p_foot = shdata + 3;

	for (int idx = 0; idx < 3 + c_nfoot; ++idx)
	{
		shdata[idx] = footinfo[idx * stride];

	}
	__syncthreads();


	float att = (val>0.0f) ? val : tex2D(texVolume, ix + 0.5f, iy + 0.5f);

	if (att == 0.0f)
		return;

	float div = att / (*p_div);
	int is = int(*p_sbin);
	for (int ifoot = 0; ifoot<c_nfoot; ++is, ++ifoot)
	{
		float area = p_foot[ifoot];

		if (is < 0 || is >= c_nr || area <= 0.0f || div == 0.0f)
			continue;

		atomicAdd(d_proj + is, div*area);
	}
}

__global__ static void apply_geodiv_kernel(float* proj)
{
	int is = blockIdx.x*blockDim.x + threadIdx.x;

	if (is >= c_nr)
		return;

	proj[is] *= tex2D(geodivTex, is + 0.5f, 0.5f);
}

//////////////////
/////// class member functions

void TAFPRAY2D::InitArealut()
{
	// setting Lines 
	int m_nlines = m_nr + 1;
	hipMalloc((void**)&cudaLines, m_nlines * sizeof(float4));
	hipMemset(cudaLines, 0, m_nlines * sizeof(float4));

	//create 2D array
	hipChannelFormatDesc chaDesc1 = hipCreateChannelDesc<float>();

	// allocate 2D array to bind 2D texture in cuda
	hipMallocArray(&d_tableArray, &chaDesc1, tableSize.x, tableSize.y);
	hipMemcpyToSymbol(HIP_SYMBOL(c_nlines), &m_nlines, sizeof(int));
}


void TAFPRAY2D::BindAreaLut()
{
	hipChannelFormatDesc channelDescLut = hipCreateChannelDesc<float>();

	// copy data to 2D array
	hipMemcpyToArray(d_tableArray, 0, 0, cudaTable, tableSize.x * tableSize.y * sizeof(float), hipMemcpyDeviceToDevice);
	areaTex.normalized = false;
	areaTex.filterMode = hipFilterModeLinear;
	areaTex.addressMode[0] = hipAddressModeClamp;
	areaTex.addressMode[1] = hipAddressModeClamp;
	// bind texture array to 2D volume
	hipBindTextureToArray(areaTex, d_tableArray, channelDescLut);
	hipBindTexture(0, lineTex, cudaLines, lineTex.channelDesc);
	float2 alutOffset = make_float2(1.0f / m_TaDeltax, 1.0f / m_TaDeltay);
	float  voxBase = fabsf(m_dx * m_dy);
	hipMemcpyToSymbol(HIP_SYMBOL(c_alutOffset), &alutOffset, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(c_voxBase), &voxBase, sizeof(float));

}

//Computing ray spreading term
void TAFPRAY2D::_cmpGeoDiv()
{
	std::vector<float>   geodiv(m_nr);
	for (int i = 0; i<m_nr; ++i)
	{
		geodiv[i] = (1.0f / m_dr);

	}
	InitGeodiv();
	BindGeodiv(&geodiv[0]);
}
//Computing Lines parameters
void TAFPRAY2D::_updateConst(float2 uv_s, float2 uv_t, float2 src, float ang)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_src), &src, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(c_uv_s), &uv_s, sizeof(float2));
	hipMemcpyToSymbol(HIP_SYMBOL(c_uv_t), &uv_t, sizeof(float2));

	ang = fmodf(ang, 360.0f);
	if (ang < 0.0f)
		ang += 360.0f;

	hipMemcpyToSymbol(HIP_SYMBOL(c_viewang), &ang, sizeof(float));

}

void TAFPRAY2D::_updateLines(float4 *cudaLines, float beta)
{
	dim3 blk(256, 1, 1);
	dim3 grd(iDivUp(m_nr + 1, blk.x), 1, 1);
	update_lines_kernel << <grd, blk >> >(cudaLines, beta);
}
// Computing pixel foot information
void TAFPRAY2D::_init_footprint(float *footinfo)
{
	dim3 blk(16, 16, 1);
	dim3 grd(iDivUp(m_nx, blk.x), iDivUp(m_ny, blk.y));
	lut_init_foot_kernel << <grd, blk >> >(footinfo);
}
//Forward projection Ax
void TAFPRAY2D::_Fp_Ax(float *d_proj, float *footinfo, const float val)
{
	// do forward projection
	_do_projection(d_proj, footinfo, val);
	// normalization by scaling the projection with geometric spreading term
	_do_proj_scaling(d_proj);
}

void TAFPRAY2D::_do_projection(float *d_proj, float *footinfo, const float val)
{
	dim3 blk(16, 16, 1);
	dim3 grd(iDivUp(m_nx, blk.x), iDivUp(m_ny, blk.y));
	lut_fp_kernel << <grd, blk >> >(d_proj, footinfo, val);
}

void TAFPRAY2D::_do_proj_scaling(float *d_proj)
{
	dim3 blk(256, 1, 1);
	dim3 grd(iDivUp(m_nr, blk.x), 1, 1);
	apply_geodiv_kernel << <grd, blk >> >(d_proj);
}




TAFPRAY2D::TAFPRAY2D(void)
{

}

TAFPRAY2D::~TAFPRAY2D(void)
{

}

void TAFPRAY2D::SetGeometry(Parameters params)
{
	m_dso = params.dso;
	m_dsd = params.dsd;
	m_nx = params.nx;
	m_ny = params.ny;
	m_dx = params.dx;
	m_dy = params.dy;
	m_offset_x = params.offset_x;
	m_offset_y = params.offset_y;
	m_nr = params.nr;
	m_dr = params.dr;
	m_offset_r = params.offset_r;
	m_angle_start = params.angle_start;
	m_na = params.na;
	m_TaDimx = params.TaDimx;
	m_TaDimy = params.TaDimy;
	m_TaDeltax = params.TaDeltax;
	m_TaDeltay = params.TaDeltay;
}

int TAFPRAY2D::DoProjection(float* h_volume, float* h_proj, float* lut_area, float* betas)
{
	// compute semi-length of image and detector
	float m_xx = m_nx * m_dx *0.5f;
	float m_yy = m_ny * m_dy *0.5f;

	float m_rr = m_nr * m_dr *0.5f;

	const int  nfoot = 5;

	// copy const variable to device
	hipMemcpyToSymbol(HIP_SYMBOL(c_dso), &m_dso, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_dsd), &m_dsd, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_nx), &m_nx, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_ny), &m_ny, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_dx), &m_dx, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_dy), &m_dy, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_offset_x), &m_offset_x, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_offset_y), &m_offset_y, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_nr), &m_nr, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_dr), &m_dr, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_offset_r), &m_offset_r, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_angle_start), &m_angle_start, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_angle_num), &m_na, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_TaDimx), &m_TaDimx, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_TaDimy), &m_TaDimy, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_TaDeltax), &m_TaDeltax, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_TaDeltay), &m_TaDeltay, sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(c_xx), &m_xx, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_yy), &m_yy, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_rr), &m_rr, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_nfoot), &nfoot, sizeof(int));

	volumeSize = make_int2(m_nx, m_ny);
	tableSize = make_int2(m_TaDimx, m_TaDimy);
	geodivSize = make_int2(m_nr, 1);

	//Computing ray spreading term and bind texture
	_cmpGeoDiv();

	//Set footprint size and set table lut
	float *footinfo;
	hipMalloc((void**)&footinfo, m_nx * m_ny * (nfoot + 3) * sizeof(float));
	hipMemset(footinfo, 0, m_nx * m_ny * (nfoot + 3) * sizeof(float));

	InitArealut();
	thrust::device_vector<float> lut_for(lut_area, lut_area + m_TaDimx * m_TaDimy);
	cudaTable = thrust::raw_pointer_cast(&lut_for[0]);
	BindAreaLut();

	//Initialization d_proj and cudavolume
	float *d_proj;
	uint size = m_nr * m_na * sizeof(float);
	hipMalloc((void **)&d_proj, size);
	hipMemset(d_proj, 0, size);

	InitProjection();
	thrust::device_vector<float> x_for(h_volume, h_volume + m_nx * m_ny);
	cudavolume = thrust::raw_pointer_cast(&x_for[0]);
	BindImgToTex2D();

	//Forward projection

	for (int ia = 0; ia < m_na; ++ia)
	{
		float beta = (betas[ia] - m_angle_start)*float(DEG2RAD);
		float   cs = cosf(beta);
		float   sn = sinf(beta);
		float2 uv_t = rotateCCW_z(make_float2(0, -1), cs, sn);
		float2 uv_s = rotateCCW_z(make_float2(1, 0), cs, sn);
		float2 src = rotateCCW_z(make_float2(0, m_dso), cs, sn);

		_updateConst(uv_s, uv_t, src, beta*float(RAD2DEG));

		//Update Lines 
		_updateLines(cudaLines, beta);

		//Update pixel foot information
		_init_footprint(footinfo);

		//Projection
		_Fp_Ax(d_proj + ia * m_nr, footinfo, -1.0f);

	}
	// Copy data to host
	hipMemcpy(h_proj, d_proj, size, hipMemcpyDeviceToHost);

	// cuda unbind
	hipUnbindTexture(texVolume);
	hipUnbindTexture(geodivTex);
	hipUnbindTexture(areaTex);
	hipUnbindTexture(lineTex);

	//Cuda free
	hipFree(footinfo);
	hipFree(d_proj);
	hipFree(cudaLines);
	hipFreeArray(d_volumeArray);
	hipFreeArray(d_tableArray);
	hipFreeArray(d_geodivArray);

	return 0;
}
